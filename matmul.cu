#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16
#define N 512

// CUDA kernel using shared memory
__global__ void MatMulShared(float *A, float *B, float *C, int n) {
    __shared__ float Asub[TILE_SIZE][TILE_SIZE];
    __shared__ float Bsub[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float sum = 0.0f;

    for (int tileIdx = 0; tileIdx < (n / TILE_SIZE); ++tileIdx) {
        if (row < n && tileIdx * TILE_SIZE + threadIdx.x < n)
            Asub[threadIdx.y][threadIdx.x] = A[row * n + tileIdx * TILE_SIZE + threadIdx.x];
        else
            Asub[threadIdx.y][threadIdx.x] = 0.0f;

        if (tileIdx * TILE_SIZE + threadIdx.y < n && col < n)
            Bsub[threadIdx.y][threadIdx.x] = B[(tileIdx * TILE_SIZE + threadIdx.y) * n + col];
        else
            Bsub[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += Asub[threadIdx.y][k] * Bsub[k][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < n && col < n)
        C[row * n + col] = sum;
}

int main() {
    int size = N * N * sizeof(float);
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    for (int i = 0; i < N * N; ++i) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 dimGrid(N / TILE_SIZE, N / TILE_SIZE);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);

    MatMulShared<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("C[0][0] = %.2f\n", h_C[0]);
    printf("C[100][100] = %.2f\n", h_C[100 * N + 100]);
    printf("C[511][511] = %.2f\n", h_C[511 * N + 511]);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
